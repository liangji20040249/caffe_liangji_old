#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/freespace_param_bottomup_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void FindBottomUpEdge(const int count,int num,int channels,int height,int width,int xstage,const Dtype * bottomdata, Dtype * topdata ) {
  CUDA_KERNEL_LOOP(index, count) {

    int v = index;

    int n = v / xstage;
    int c = 0;
    int xstep = width / (xstage-1);
    int current_stage = index % xstage;

    int w = current_stage * xstep;

    if(current_stage == xstage -1)
    {
      w = width -1;
    }

    int samplecount = channels * height * width;
    int spcount = height * width;

    int h=0;
    bool findedge = false;
    Dtype value = height-1;
    for(h=height-1;h>0;h-=1)
    {


      if(bottomdata[n*samplecount + c * spcount + h * width + w] ==1 &&  bottomdata[n*samplecount + c * spcount + (h-1) * width + w]==0)
      {
        findedge = true;
        value = h;
        break;
      }
    }
    topdata[n * xstage + current_stage] = height -1 - value;
  }
}


template <typename Dtype>
__global__ void FindBottomUpEdge_TowDim(const int count,int num,int channels,int height,int width,int xstage,int ystage,const Dtype * bottomdata, Dtype * topdata ) {
  CUDA_KERNEL_LOOP(index, count) {

    int v = index;

    int n = v / xstage;
    int c = 0;
    int xstep = width / (xstage-1);
    int current_stage = index % xstage;

    int w = current_stage * xstep;

    if(current_stage == xstage -1)
    {
      w = width -1;
    }

    int samplecount = channels * height * width;
    int spcount = height * width;

    int h=0;
    bool findedge = false;
    Dtype value = height-1;
    for(h=height-1;h>0;h-=1)
    {


      if(bottomdata[n*samplecount + c * spcount + h * width + w] ==1 &&  bottomdata[n*samplecount + c * spcount + (h-1) * width + w]==0)
      {
        findedge = true;
        value = h;
        break;
      }
    }

    int current_stage_x = current_stage;

    int k=0;
    int src_pos_y = 0;
    
    for(k=0;k<ystage;k++)
    {
      src_pos_y = float(k) * float(height) / float(ystage -1);;
      if(k == ystage -1)
      {
        src_pos_y = height -1;
      }
    
      topdata[n*1*xstage*ystage + k * xstage + current_stage_x] = value - src_pos_y;

    }

  }
}

template <typename Dtype>
void FreespaceParamBottomUpLayer<Dtype>::Forward_gpu_onedim(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {

        int num = bottom[0]->num();
        int channels = bottom[0]->channels();
        int height = bottom[0]->height();
        int width = bottom[0]->width();
        int xstage = this->layer_param_.freespace_param_bottomup_param().xstage();
        const Dtype * bottomdata = bottom[0]->gpu_data();
        Dtype * topdata = top[0]->mutable_gpu_data();

        const int count = num * xstage;



        FindBottomUpEdge<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count, num,channels,height,width,xstage,bottomdata,topdata);
        CUDA_POST_KERNEL_CHECK;

}

template <typename Dtype>
void FreespaceParamBottomUpLayer<Dtype>::Forward_gpu_twodim(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {

        int num = bottom[0]->num();
        int channels = bottom[0]->channels();
        int height = bottom[0]->height();
        int width = bottom[0]->width();
        int xstage = this->layer_param_.freespace_param_bottomup_param().xstage();
	int ystage = this->layer_param_.freespace_param_bottomup_param().ystage();
        const Dtype * bottomdata = bottom[0]->gpu_data();
        Dtype * topdata = top[0]->mutable_gpu_data();

        const int count = num * xstage;



        FindBottomUpEdge_TowDim<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count, num,channels,height,width,xstage,ystage,bottomdata,topdata);
        CUDA_POST_KERNEL_CHECK;

}

template <typename Dtype>
void FreespaceParamBottomUpLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {

        switch (this->layer_param_.freespace_param_bottomup_param().type())
        {
                  case FreespaceParamBottomUpParameter_Type_ONEDIM:
                        this->Forward_gpu_onedim(bottom,top);
                        break;
                  case FreespaceParamBottomUpParameter_Type_TWODIM:
                        this->Forward_gpu_twodim(bottom,top);
                        break;
                  default:
                        LOG(FATAL)<<"unknown type";
                        break;

        }

}

template <typename Dtype>
void FreespaceParamBottomUpLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
        this->Backward_gpu(top,propagate_down,bottom);
}

INSTANTIATE_LAYER_GPU_FUNCS(FreespaceParamBottomUpLayer);

}  // namespace caffe
