#include "hip/hip_runtime.h"
/*
 * Author: Liangji 
 * Email: liangji20040249@gmail.com
*/
#include <vector>

#include "caffe/blob.hpp"
#include "caffe/common.hpp"
#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/layers/generateseglabel_layer.hpp"
#include "caffe/util/io.hpp"

namespace caffe {

template <typename Dtype>
__device__ Dtype get_data(Dtype * data, int num, int channels,int height, int width,int n,int c,int h,int w,float v)
{
	if(h<0 || h >=height)
		return Dtype(v);
	if(w<0 || w >= width)
		return Dtype(v);
	
	return data[n*channels*height*width + c * height*width + h * width + w];
}
template <typename Dtype>
__device__ void set_data(Dtype * data, int num, int channels,int height, int width,int n,int c,int h,int w,Dtype v)
{
	if(h<0 || h >=height)
		return ;
	if(w<0 || w >= width)
		return ;
	
	data[n*channels*height*width + c * height*width + h * width + w]=v;
}



template <typename Dtype>
__global__ void get_seglabel(const int count, const Dtype* bottom_data,Dtype * top1_data,Dtype * top2_data,Dtype * top3_data,Dtype * top4_data,Dtype * top5_data,
				int num,int in_height,int in_width,int out_channels,int out_height,int out_width,int ignore_label,int seg_ignore_range,
				int reg_range,float resize_ratio,int seg_label_shift) {
CUDA_KERNEL_LOOP(index, count) {


	int temp=index;

	int n,c,h,w;
	n = temp / (out_height * out_width);
	temp = temp % (out_height * out_width);
	h = temp / out_width;
	temp = temp % out_width;
	w = temp;
	
	int sn,sc,sh,sw;
	sn=n;
	sc=0;
	
	sh = float(h) * resize_ratio;
	sw = float(w) * resize_ratio;

	bool isedge=false;
	//Dtype srcv = get_data(bottom_data, num, 1,in_height, in_width,sn,sc,sh,sw,float(ignore_label));
	//Dtype srcv = bottom_data[sn*1*in_height*in_width + sc * in_height*in_width + sh * in_width + sw];
	
	int idx = sn*1*in_height*in_width + sc * in_height*in_width + sh * in_width + sw;
	Dtype srcv = ignore_label;
	if(sw<0||sw>=in_width||sh<0||sh>=in_height)
	{
		;
	}
	else if(idx>=0 && idx < num * 1 * in_height * in_width)
		srcv = bottom_data[idx];


	if(int(srcv)==ignore_label)
	{
		set_data(top2_data, num, 1,out_height, out_width, n, 0, h, w,Dtype(0));
		return;
	}

	Dtype nebv;
	float min_dist = reg_range*2;
	float min_dx,min_dy;
	float dist;
	for(int i=-reg_range;i<=reg_range;i++)
	{
		for(int j=-reg_range;j<=reg_range;j++)
		{
			
if(i==0 && j ==0)
			{
				continue;
			}
			int x=sw+i;
			int y=sh+j;
			if(x<0||x>=in_width||y<0||y>=in_height)
			{
				continue;
			}
			
			nebv = srcv;
			idx = sn*1*in_height*in_width + sc * in_height*in_width + y * in_width + x;
			if(idx>=0 && idx < num * in_height * in_width)
				nebv = bottom_data[idx];
			

			//nebv = get_data(bottom_data, num, 1,in_height, in_width,sn,sc,y,x,float(srcv));
			//nebv = bottom_data[sn*1*in_height*in_width + sc * in_height*in_width + y * in_width + x];
			if(int(srcv)!=int(nebv))
			{
				isedge = true;
				dist = sqrt(float(i*i + j*j));
				if(dist < min_dist)
				{
					min_dist = dist;
					min_dx = i;
					min_dy = j;
				}
			}
		}
	}


	
	//set expand seg label
	int lb = (int)srcv - seg_label_shift;
	if(lb>=0 && lb < out_channels)
	{
	
		//top1_data[0]=1.0;
		//top1_data[n*out_channels*out_height*out_width + lb * out_height*out_width + h * out_width + w]=1;
		//top1_data[n*out_channels*out_height*out_width + 0 * out_height*out_width + h * out_width + w]=1;
		
		set_data(top1_data, num, out_channels,out_height, out_width, n, lb, h, w,Dtype(1));
	}

	//set seg ignore
	if(isedge && min_dist <= seg_ignore_range)
	{
		set_data(top2_data, num, 1,out_height, out_width, n, 0, h, w,Dtype(0));
	}
	/*if(int(srcv) == ignore_label)
	{
		set_data(top2_data, num, 1,out_height, out_width, n, 0, h, w,Dtype(0));
	}*/

	//set reg and mask
	if(isedge && min_dist <= reg_range)
	{
		set_data(top3_data, num, 1,out_height, out_width, n, 0, h, w,Dtype(min_dist));
		set_data(top5_data, num, 2,out_height, out_width, n, 0, h, w,Dtype(min_dx));
		set_data(top5_data, num, 2,out_height, out_width, n, 1, h, w,Dtype(min_dy));
		set_data(top4_data, num, 1,out_height, out_width, n, 0, h, w,Dtype(1));	
	}


}
}

template <typename Dtype>
void GenerateSeglabelLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
    


	//H_.Reshape(num_, channels_, height_, width_);
	//caffe_copy(H_.count(),bottom[0]->gpu_data(),H_.mutable_gpu_data());
	//const Dtype* bottom_data = H_.gpu_data();
	const Dtype* bottom_data = bottom[0]->gpu_data();
	//Dtype* bottom_data = bottom[0]->mutable_gpu_data();
	Dtype* top1_data = top[0]->mutable_gpu_data();
	Dtype* top2_data = top[1]->mutable_gpu_data();
	Dtype* top3_data = top[2]->mutable_gpu_data();
	Dtype* top4_data = top[3]->mutable_gpu_data();
	Dtype* top5_data = top[4]->mutable_gpu_data();
	

	caffe_gpu_set(top[0]->count(), Dtype(0),top1_data);
	caffe_gpu_set(top[1]->count(), Dtype(1),top2_data);
	caffe_gpu_set(top[2]->count(), Dtype(0),top3_data);
	caffe_gpu_set(top[3]->count(), Dtype(0),top4_data);
	caffe_gpu_set(top[4]->count(), Dtype(0),top5_data);

	const int count = num_ * out_height_ * out_width_;

	
	get_seglabel<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count,bottom_data,top1_data,top2_data,top3_data,top4_data,top5_data,num_,height_,width_,out_channels_,out_height_,out_width_,ignore_label_,seg_ignore_range_,reg_range_,resize_ratio_,seg_label_shift_);

	CUDA_POST_KERNEL_CHECK;
	


}

template <typename Dtype>
void GenerateSeglabelLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {

    	;

}

INSTANTIATE_LAYER_GPU_FUNCS(GenerateSeglabelLayer);

}  // namespace caffe
