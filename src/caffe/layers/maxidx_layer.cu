#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/maxidx_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__device__ Dtype get_data(Dtype * data, int num, int channels,int height, int width,int n,int c,int h,int w)
{
	if(h<0 || h >=height)
		return 0;
	if(w<0 || w >= width)
		return 0;
	
	return data[n*channels*height*width + c * height*width + h * width + w];
}

template <typename Dtype>
__device__ void set_data(Dtype * data, int num, int channels,int height, int width,int n,int c,int h,int w,Dtype v)
{
	if(h<0 || h >=height)
		return ;
	if(w<0 || w >= width)
		return ;
	

	data[n*channels*height*width + c * height*width + h * width + w]=v;
}




template <typename Dtype>
__global__ void forward_idx(const int count, const Dtype * bottom_data,Dtype * top_data,int num,int channels,int height,int width,int sub_channels,int idx_num)
{
CUDA_KERNEL_LOOP(index, count) {

	
	int n,c,h,w;
	int temp=index;

	n = temp / (height*width);
	temp = temp % (height*width);
	h = temp / width;
	temp = temp % width;
	w = temp;

	int idx=-1;
	Dtype maxvalue=0;
	Dtype curvalue=0;

	for(int i=0;i<idx_num;i++)
	{
		curvalue = 0;
		for(int j=0;j<sub_channels;j++)
		{
			Dtype v = get_data(bottom_data,  num,  channels, height,  width, n,i*sub_channels + j, h, w);
			curvalue = curvalue + v*v;
		}
		if(idx<0)
		{
			idx = 0;
			maxvalue = curvalue;
		}
		else
		{
			if(curvalue > maxvalue)
			{
				idx = i;
				maxvalue = curvalue;
			}
		}
	}
	set_data(top_data,  num,  idx_num, height,  width, n, idx, h, w,Dtype(1.0));
}

}

template <typename Dtype>
void MaxIdxLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  
	

	const int count = height_ * width_ * num_;
	const Dtype * bottom_data = bottom[0]->gpu_data();
	Dtype * top_data = top[0]->mutable_gpu_data();
	int sub_channels = channels_ / idx_num_;

	caffe_gpu_set(top[0]->count(),Dtype(0),top_data);
	
	
	forward_idx<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count,bottom_data,top_data,num_,channels_,height_,width_,sub_channels,idx_num_);

	CUDA_POST_KERNEL_CHECK;
	
}

template <typename Dtype>
void MaxIdxLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
 ;
}

INSTANTIATE_LAYER_GPU_FUNCS(MaxIdxLayer);

}  // namespace caffe
