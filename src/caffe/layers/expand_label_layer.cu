#include "hip/hip_runtime.h"
/*
 * Author: Liangji 
 * Email: liangji20040249@gmail.com
*/
#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/im2col.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/layers/expand_label_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void cal_select_one_label(const int count, Dtype select_one_label,int num,int outchannels,int height,int width,const Dtype * bottom_data,Dtype * top_data) 
{
      CUDA_KERNEL_LOOP(index, count) {
            int v = index;
	    int topdim = height*width*2;
            int spdim = height * width;
            int n,c,h,w;
            c=0;

            n = v / spdim;
            v = v % spdim;
            h = v / width;
            v = v % width;
            w = v;

            Dtype value = bottom_data[n*spdim + 0 + h*width + w];
	    if(int(value) != int(select_one_label))
            {
                  top_data[n*topdim + int(0)*spdim + h*width + w] = 1;
            }
            else
            {
                  top_data[n*topdim + int(1)*spdim + h*width + w] = 1;
            }
            
      }
}

template <typename Dtype>
__global__ void cal_select_one_label_withignore(const int count, Dtype select_one_label,int num,int outchannels,int height,int width,const Dtype * bottom_data,Dtype * top_data,Dtype * ignore_data, Dtype ignore_label) 
{
      CUDA_KERNEL_LOOP(index, count) {
            int v = index;
	    int topdim = height * width *2;
            int spdim = height * width;
            int n,c,h,w;
            c=0;

            n = v / spdim;
            v = v % spdim;
            h = v / width;
            v = v % width;
            w = v;

            Dtype value = bottom_data[n*spdim + 0 + h*width + w];

            if(int(value) == int(ignore_label))
            {
                  ignore_data[n*spdim + 0 + h*width + w] = 0;
                  top_data[n*topdim + int(0)*spdim + h*width + w] = 1;
            }
            else
            {
            if(int(value) != int(select_one_label))
            {
                  top_data[n*topdim + int(0)*spdim + h*width + w] = 1;
            }
            else
            {
                  top_data[n*topdim + int(1)*spdim + h*width + w] = 1;
            }
            }


      }
}


template <typename Dtype>
__global__ void cal_expand_label(const int count, int num,int outchannels,int height,int width,const Dtype * bottom_data,Dtype * top_data) 
{
      CUDA_KERNEL_LOOP(index, count) {
            int v = index;
            int spdim = height * width;
            int topdim = height * width * outchannels;
            int n,c,h,w;
            c=0;

            n = v / spdim;
            v = v % spdim;
            h = v / width;
            v = v % width;
            w = v;

            Dtype value = bottom_data[n*spdim + 0 + h*width + w];
	    if(value < 0 || value >= outchannels)
		return;
            top_data[n*topdim + int(value)*spdim + h*width + w] = 1;
      }
}

template <typename Dtype>
__global__ void cal_expand_label_withignore(const int count, int num,int outchannels,int height,int width,const Dtype * bottom_data,Dtype * top_data,Dtype * ignore_data, Dtype ignore_label) 
{
      CUDA_KERNEL_LOOP(index, count) {
            int v = index;
            int spdim = height * width;
	    int topdim = height * width * outchannels;
            int n,c,h,w;
            c=0;

            n = v / spdim;
            v = v % spdim;
            h = v / width;
            v = v % width;
            w = v;

            Dtype value = bottom_data[n*spdim + 0 + h*width + w];

	    if(int(value) == int(ignore_label))
	    {
		ignore_data[n*spdim + 0 + h*width + w] = 0;
	    }
	    else
            {
	      if(value < 0 || value >= outchannels)
                  ;
              else
                  top_data[n*topdim + int(value)*spdim + h*width + w] = 1;
            }
      }
}


template <typename Dtype>
void ExpandLabelLayer<Dtype>::Forward_expand_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {

      int num = bottom[0]->num();
      int channels = bottom[0]->channels();
      int height = bottom[0]->height();
      int width = bottom[0]->width();
      const Dtype* bottom_data = bottom[0]->gpu_data();
      Dtype* top_data = top[0]->mutable_gpu_data();
      
      int outchannels = top[0]->channels();

      caffe_gpu_set(top[0]->count(),Dtype(0),top_data);


      const int count = bottom[0]->num() * bottom[0]->height() * bottom[0]->width();

      if(top.size() == 1)
      {
            cal_expand_label<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count, num,outchannels,height,width,bottom_data,top_data);
            CUDA_POST_KERNEL_CHECK;
      }
      else
      {
            Dtype* ignore_data = top[1]->mutable_gpu_data();
            caffe_gpu_set(top[1]->count(),Dtype(1),ignore_data);

            cal_expand_label_withignore<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count, num,outchannels,height,width,bottom_data,top_data,ignore_data,ignore_label_);
            CUDA_POST_KERNEL_CHECK;
      }

}


template <typename Dtype>
void ExpandLabelLayer<Dtype>::Forward_select_one_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {

      int num = bottom[0]->num();
      int channels = bottom[0]->channels();
      int height = bottom[0]->height();
      int width = bottom[0]->width();
      const Dtype* bottom_data = bottom[0]->gpu_data();
      Dtype* top_data = top[0]->mutable_gpu_data();
      
      int outchannels = top[0]->channels();

      caffe_gpu_set(top[0]->count(),Dtype(0),top_data);

      //int sp_dim = bottom[0]->count() / bottom[0]->num(); 
      int sp_dim = bottom[0]->height() * bottom[0]->width();

      Dtype select_one_label = this->layer_param_.expand_label_param().select_one_label();

      const int count = bottom[0]->num() * sp_dim;
      if(top.size() == 1)
      {
            cal_select_one_label<Dtype><<<CAFFE_GET_BLOCKS(count),CAFFE_CUDA_NUM_THREADS>>>(count, select_one_label,num,outchannels,height,width,bottom_data,top_data);
            CUDA_POST_KERNEL_CHECK;
      }
      else
      {
            Dtype* ignore_data = top[1]->mutable_gpu_data();
            caffe_gpu_set(top[1]->count(),Dtype(1),ignore_data);

            cal_select_one_label_withignore<Dtype><<<CAFFE_GET_BLOCKS(count),CAFFE_CUDA_NUM_THREADS>>>(count, select_one_label,num,outchannels,height,width,bottom_data,top_data,ignore_data,ignore_label_);
            CUDA_POST_KERNEL_CHECK;
      }

}

template <typename Dtype>
void ExpandLabelLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {

      int num = bottom[0]->num();
      int channels = bottom[0]->channels();
      int height = bottom[0]->height();
      int width = bottom[0]->width();
      const Dtype* bottom_data = bottom[0]->gpu_data();
      Dtype* top_data = top[0]->mutable_gpu_data();
      
      int outchannels = top[0]->channels();

      caffe_gpu_set(top[0]->count(),Dtype(0),top_data);

      int sp_dim = bottom[0]->count() / bottom[0]->num(); 

      

      switch (this->layer_param_.expand_label_param().type())
      {
            case ExpandLabelParameter_Type_EXPAND:
                  this->Forward_expand_gpu(bottom,top);
                  break;
            case ExpandLabelParameter_Type_SELECT_ONE:
                  this->Forward_select_one_gpu(bottom,top);
                  break;
            default:
                  LOG(FATAL) << "Unknown type method.";

      }

}

/// @brief refer to CPU backward -- the BLAS implementation is the same.
template <typename Dtype>
void ExpandLabelLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {

		CHECK(false);
  
}


INSTANTIATE_LAYER_GPU_FUNCS(ExpandLabelLayer);

}  // namespace caffe
