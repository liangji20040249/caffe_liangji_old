#include "hip/hip_runtime.h"
/*
 * Author: Liangji 
 * Email: liangji20040249@gmail.com
*/
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/io.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/layers/simplehardsample_layer.hpp"

namespace caffe {


template <typename Dtype>
__global__ void back_diff(const int count, const Dtype * topdiff, Dtype * bottomdiff,Dtype thres) {
CUDA_KERNEL_LOOP(index, count) {
	Dtype v = topdiff[index];
	if(v >= thres || v <= -thres)
		bottomdiff[index]=v;
	else
		bottomdiff[index]=0;


}
}



template <typename Dtype>
void SimpleHardSampleLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
    if(bottom.size()>2)
    {
        caffe_gpu_mul(bottom[0]->count(), bottom[0]->gpu_data(),bottom[2]->gpu_data(), top[0]->mutable_gpu_data());
    }
    else
    {
        caffe_copy(bottom[0]->count(), bottom[0]->gpu_data(), top[0]->mutable_gpu_data());
    }
}


template <typename Dtype>
void SimpleHardSampleLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {

      switch (this->layer_param_.simplehardsample_param().type())
      {
            case SimpleHardSampleParameter_Type_SORT:
                  this->Backward_gpu_sort(top,propagate_down,bottom);
                  break;
            case SimpleHardSampleParameter_Type_MEAN:
                  this->Backward_gpu_mean(top,propagate_down,bottom);
                  break;
            default:
                  LOG(FATAL) << "Unknown type method.";

      }


}

template <typename Dtype>
void SimpleHardSampleLayer<Dtype>::Backward_gpu_mean(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {

	const Dtype * top_diff = top[0]->gpu_diff();
	Dtype * bottom_diff = bottom[0]->mutable_gpu_diff();
	const int count = top[0]->count();
	

	Dtype asum=0;
	Dtype * vp = &asum;
	caffe_gpu_asum(count, top_diff,vp);
	
	Dtype meandiff = 0;
	if(count > 0)
	{
		meandiff = asum / Dtype(count);
	}
	//LOG(INFO)<<"top_diff mean:"<<meandiff;

	back_diff<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count,top_diff,bottom_diff,meandiff);
	CUDA_POST_KERNEL_CHECK;


	

}

template <typename Dtype>
void SimpleHardSampleLayer<Dtype>::Backward_gpu_sort(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
    
	

	if(std::fabs(remain_hard_rate_ -1.0) < FLT_EPS_)
	{
		caffe_copy(bottom[0]->count(),top[0]->gpu_diff(),bottom[0]->mutable_gpu_diff());
		return;
	}

	const Dtype * topdiff = top[0]->cpu_diff();
	sorted_top_diff_.clear();
	Dtype v;
	for(int i=0;i<top[0]->count();i++)
	{
		v = std::fabs(topdiff[i]);
		if(v > FLT_EPS_)
			sorted_top_diff_.push_back(v);
	}
	if(sorted_top_diff_.size()<1)
		return;

	std::sort(sorted_top_diff_.begin(),sorted_top_diff_.end());
	
	int idx = (1-remain_hard_rate_) * float(sorted_top_diff_.size());
	Dtype thres = sorted_top_diff_[idx];
	
	//LOG(INFO)<<"hardrate:"<<remain_hard_rate_<<", thres:"<<thres<<", idx:"<<idx<<", sort diff size:"<<sorted_top_diff_.size();

	const Dtype * top_diff = top[0]->gpu_diff();
	Dtype * bottom_diff = bottom[0]->mutable_gpu_diff();
	const int count = bottom[0]->count();
	back_diff<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count,top_diff,bottom_diff,thres);
	CUDA_POST_KERNEL_CHECK;

}

INSTANTIATE_LAYER_GPU_FUNCS(SimpleHardSampleLayer);

}  // namespace caffe
